#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<hipfft/hipfft.h>
#include<hipblas.h>
#include<hipblas.h>

#include <iostream>
#include"registrationGPU.h"


#define pi (atan(1.0)*4)

inline int iDivUp(int a, int b) {
	return (a%b != 0) ? (a / b + 1) : (a / b);
}

struct IndexOffset
{
	IndexOffset(unsigned int num)
	{
		indexX = new float[num]();
		indexY = new float[num]();
	}
	~IndexOffset()
	{
		if (indexX != NULL) { delete[] indexX; indexX = NULL;
		}
		if (indexY != NULL) { delete[] indexY; indexY = NULL;
		}
	}

	float *indexY;//��
	float *indexX;//��
};

void fft2(float *d_input,hipfftComplex *d_fftResult, unsigned int height, unsigned int width,unsigned int num)
{
	//��άfft��������
	int n[2] = { height,width };
	int inembed[] = { height,width };
	int onembed[] = { height,width / 2 + 1 };
	hipfftHandle fftPlanFwd;

	//������FFT�ľ������
	hipfftPlanMany(&fftPlanFwd, 2, n, inembed, 1, height*width, onembed, 1, height*(width / 2 + 1), HIPFFT_R2C, num);

	hipfftExecR2C(fftPlanFwd, d_input, d_fftResult);

	hipfftDestroy(fftPlanFwd);
}

__global__ void ifftshiftKernelstep1(float *d_src, int length,int begin)
{
	unsigned int ix = threadIdx.x + blockIdx.x*blockDim.x;
	if (ix < length)
	{
		d_src[ix] = ix + begin;
	}
}

__global__ void ifftshiftKernelstep2(float *d_src, float *d_dst, int length)
{
	unsigned int ix = threadIdx.x + blockIdx.x*blockDim.x;
	if (ix < length)
	{
		int half = ceilf((float)length / 2);
		if (ix < half)
		{
			d_dst[ix] = d_src[ix + half];
		}
		else
		{
			d_dst[ix] = d_src[ix - half];
		}
	}
}

void ifftshiftVecGPU(float *d_dst, int begin, int end)
{
	int length = end - begin + 1;

	float *temp = NULL; hipMalloc(&temp, sizeof(float)*length);

	ifftshiftKernelstep1 << <iDivUp(length, 128), 128 >> > (temp, length, begin);//��ֵ
	ifftshiftKernelstep2 << <iDivUp(length, 128), 128 >> > (temp, d_dst, length);//���ҶԵ�

	hipFree(temp);
}

template<class T>
void ifftshiftVec(T *dst, int begin, int end)
{
	int ii = 0;
	for (int i = 0; i <= end; ++i)
	{
		dst[ii] = i;
		ii++;
	}
	for (int i = begin; i < 0; ++i)
	{
		dst[ii] = i;
		ii++;
	}
}

inline __device__ void mulComplexConj(hipfftComplex &a, hipfftComplex &b, const float &n)
{
	hipfftComplex temp;
	temp.x = (a.x*b.x + a.y*b.y)*n;
	temp.y = (-a.x*b.y + b.x*a.y)*n;
	b = temp;
}

__global__ void dot(hipfftComplex *d_A, hipfftComplex *d_B,hipDoubleComplex *d_C ,unsigned int dataSize,int num, float n)
{
	size_t i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < dataSize)
	{
		hipfftComplex a = d_A[i];
		for (int k = 0; k < num; ++k)
		{
			hipfftComplex b = d_B[i + k * dataSize];
			mulComplexConj(a, b, n);
			d_C[i + k * dataSize] = hipComplexFloatToDouble(b);
		}
	}
}

__global__ void dot1(hipfftComplex *d_A, hipfftComplex *d_B, hipfftComplex *d_C, unsigned int dataSize, int num, float n)
{
	size_t i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < dataSize)
	{
		for (int k = 0; k < num; ++k)
		{
			hipfftComplex a = d_A[i + k * dataSize];
			hipfftComplex b = d_B[i];
			mulComplexConj(a, b, n);
			d_C[i + k * dataSize] = b;
		}
	}
}

__global__ void getImFTout(hipDoubleComplex *imFTout, hipDoubleComplex *imFT, unsigned int inputH,
	unsigned int inputW, unsigned int outputH, unsigned int outputW,int Ystart)
{
	unsigned int ix = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y*blockDim.y;
	if (ix < inputW&&iy < inputH)
	{
		int halfH = ceilf((float)inputH / 2);
		if (iy <= halfH)
		{
			imFTout[ix + iy * outputW] = imFT[ix + iy * inputW];
		}
		else
		{
			imFTout[ix + (Ystart + iy)*outputW] = imFT[ix + iy * inputW];
		}
		
	}
}

void FTpad(hipDoubleComplex *imFTout, hipDoubleComplex *imFT,unsigned int inputH,unsigned int inputW, unsigned int outputH, unsigned int outputW,int num)
{
	int Ystart = outputH - inputH;
	dim3 block(16, 16);
	dim3 grid(iDivUp(inputW, block.x), iDivUp(inputH, block.y));
	for (int i = 0; i < num; ++i)
	{
		getImFTout << <grid, block >> > (imFTout+i*outputH*outputW, imFT+i*inputH*inputW, inputH, inputW, outputH, outputW, Ystart);
	}
}

void ifft2(hipDoubleComplex *d_output, hipDoubleComplex *d_fftResult, unsigned int height, unsigned int width, unsigned int num)
{
	//��άifft��������
	int n[2] = { height,width };
	int inembed[] = { height,width };
	//int onembed[] = { height,width / 2 + 1 };
	hipfftHandle fftPlanInv;


	hipfftPlanMany(&fftPlanInv, 2, n, inembed, 1,height*width, inembed, 1, height*width, HIPFFT_Z2Z,num);
	hipfftExecZ2Z(fftPlanInv, d_fftResult, d_output, HIPFFT_BACKWARD);//HIPFFT_BACKWARD
	//hipfftPlanMany(&fftPlanInv, 2, n, onembed, 1, height*(width / 2 + 1), onembed, 1, height*(width / 2 + 1), HIPFFT_Z2Z, num);
	//hipfftExecZ2Z(fftPlanInv, d_fftResult, d_fftResult, HIPFFT_BACKWARD);//HIPFFT_BACKWARD
	
	hipfftDestroy(fftPlanInv);
}

template<class T>
__global__ void getIn(T *d_dot,T *in,unsigned int height,unsigned int width)
{
	unsigned int ix = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y*blockDim.y;//nc
	if (ix < width&&iy < height)
	{
		if (ix > width/2 && ix < width)
		{
			if (iy == 0)
			{
				in[ix + iy * width].x = d_dot[(width - ix) + iy * (width / 2 + 1)].x;
				in[ix + iy * width].y = -d_dot[(width - ix) + iy * (width / 2 + 1)].y;
			}
			else
			{
				in[ix + iy * width].x = d_dot[(width - ix) + (height - iy) * (width / 2 + 1)].x;
				in[ix + iy * width].y = -d_dot[(width - ix) + (height - iy) * (width / 2 + 1)].y;
			}
		}
		else
		{
			in[ix + iy * width] = d_dot[ix + iy * (width / 2 + 1)];
		}
	}
}

void getDftupsIn(hipfftComplex *buf1ft, hipfftComplex *buf2ft, hipfftComplex *in, unsigned int height, unsigned int width)
{
	int halflength = height * (width / 2 + 1);
	hipfftComplex *d_dot; hipMalloc(&d_dot, sizeof(hipfftComplex)*halflength);//!
	dot1 << <iDivUp(halflength, 128), 128 >> > (buf2ft, buf1ft, d_dot, halflength, 1, 1);
	//������չ
	dim3 block(16, 16);
	dim3 grid(iDivUp(width, block.x), iDivUp(height, block.y));
	getIn << <grid, block >> > (d_dot, in, height, width);

	hipFree(d_dot);
}

template<class T,class R>
__global__ void VecMiusConst(T *input, R data, unsigned int length)
{
	unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < length)
	{
		input[i] = input[i] - data;
	}
}

__global__ void getNocVec(float *nocVec, float coff, unsigned int length)
{
	unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < length)
	{
		nocVec[i] = i - coff;
	}
}

__global__ void getkernc(hipfftComplex *kernc, float *temp,int nc,float usfac, unsigned int length)
{
	unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < length)
	{
		kernc[i].x = cos((2 * pi / (nc*usfac))* temp[i]);
		kernc[i].y = -sin((2 * pi / (nc*usfac))* temp[i]);
	}
}

void computeKernc(hipfftComplex *kernc, hipblasHandle_t handle, int nc, int noc, float coff, float usfac)
{
	float *xita1; hipMalloc(&xita1, sizeof(float)*nc);//!
	ifftshiftVecGPU(xita1, 0, nc - 1);
	VecMiusConst<float, float> << <iDivUp(nc, 128), 128 >> > (xita1, floor(nc / 2), nc);
	float *xita2; hipMalloc(&xita2, sizeof(float)*noc);//!
	getNocVec << <iDivUp(noc, 128), 128 >> > (xita2, coff, noc);

	//�������
	float a = 1, b = 0;
	//(NC*1)  *  (1*NOC)
	float *temp; hipMalloc(&temp, sizeof(float)*noc*nc);//!
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, noc, nc, 1, &a, xita2, noc, xita1, 1, &b, temp, noc);

	getkernc << <iDivUp((int)(nc*noc), 128), 128 >> > (kernc, temp, nc, usfac, nc*noc);

	hipFree(xita1); hipFree(xita2);
	hipFree(temp);
}

void computeKernr(hipfftComplex *kernr, hipblasHandle_t handle, int nr, int nor, float roff, float usfac)
{
	float *xita2; hipMalloc(&xita2, sizeof(float)*nr);//!
	ifftshiftVecGPU(xita2, 0, nr - 1);
	VecMiusConst<float, float> << <iDivUp(nr, 128), 128 >> > (xita2, floor(nr / 2), nr);
	float *xita1; hipMalloc(&xita1, sizeof(float)*nor);//!
	getNocVec << <iDivUp(nor, 128), 128 >> > (xita1, roff, nor);


	//�������
	float a = 1, b = 0;
	//(nor*1)  *  (1*nr)
	float *temp; hipMalloc(&temp, sizeof(float)*nor*nr);//!
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nr, nor, 1, &a, xita2, nr, xita1, 1, &b, temp, nr);
	getkernc << <iDivUp((int)(nr*nor), 128), 128 >> > (kernr, temp, nr, usfac, nr*nor);

	hipFree(xita1); hipFree(xita2);
	hipFree(temp);
}

__global__ void ComplexFloat2Double(hipfftComplex *input, hipDoubleComplex *output, unsigned int length)
{
	unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < length)
	{
		output[i] = hipComplexFloatToDouble(input[i]);
	}
}

void getDftupsOut(hipfftComplex *kernr, hipfftComplex *in, hipfftComplex *kernc, hipDoubleComplex *CCDouble, hipblasHandle_t handle, int nor, int noc, int nr,int nc)
{
	//����ת��
	hipDoubleComplex *tempDouble; hipMalloc(&tempDouble, sizeof(hipDoubleComplex)*nor*nc);//!
	hipDoubleComplex *kerncDouble; hipMalloc(&kerncDouble, sizeof(hipDoubleComplex)*nc*noc);//!
	hipDoubleComplex *kernrDouble; hipMalloc(&kernrDouble, sizeof(hipDoubleComplex)*nr*nor);//!
	hipDoubleComplex *inDouble; hipMalloc(&inDouble, sizeof(hipDoubleComplex)*nc*nr);//!

	ComplexFloat2Double << <iDivUp(noc*nc, 128), 128 >> > (kernc, kerncDouble, noc*nc);
	ComplexFloat2Double << <iDivUp(nor*nr, 128), 128 >> > (kernr, kernrDouble, nor*nr);
	ComplexFloat2Double << <iDivUp(nr*nc, 128), 128 >> > (in, inDouble, nr*nc);
	//hipfftComplex *temp; hipMalloc(&temp, sizeof(hipfftComplex)*nor*nc);
	hipDoubleComplex a; a.x = 1; a.y = -0;
	hipDoubleComplex b; b.x = 0; b.y = 0;

	//1 (nor*nr)*(nr*nc)=(nor*nc)
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nc, nor, nr, &a, inDouble, nc, kernrDouble, nr, &b, tempDouble, nc);

	//2 (nor*nc)*(nc*noc)=(nor*noc) nor=72 noc=75 nor=74 noc=75 (temp:nor kernc:noc)
	hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, noc, nor, nc, &a, kerncDouble, noc, tempDouble, nc, &b, CCDouble, noc);

	hipFree(tempDouble); hipFree(kerncDouble); 
	hipFree(kernrDouble); hipFree(inDouble);
}

__global__ void Cabs(hipDoubleComplex *CC, double *CCabs, unsigned int length)
{
	unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < length)
	{
		//CCabs[i] = sqrt((double)CC[i].x*(double)CC[i].x + (double)CC[i].y*(double)CC[i].y);
		CCabs[i] = hipCabs(CC[i]);
	}
}

void dftups(double *CCabs, hipfftComplex *in, hipblasHandle_t handle, int nor, int noc, float usfac, float roff, float coff,unsigned int height,unsigned int width)
{
	//ŷ����ʽ exp(i��)=cos(��)+i*sin(��);
	//�����
	int nr = height;
	int nc = width;
	hipDoubleComplex *CC; hipMalloc(&CC, sizeof(hipDoubleComplex)*nor*noc);//!

	hipfftComplex *kernc; hipMalloc(&kernc, sizeof(hipfftComplex)*noc*nc);//!
	computeKernc(kernc,handle, nc, noc, coff, usfac);
	hipfftComplex *kernr; hipMalloc(&kernr, sizeof(hipfftComplex)*nor*nr);//!
	computeKernr(kernr,handle, nr, nor, roff, usfac);
	//��CC
	getDftupsOut(kernr, in, kernc, CC,handle, nor, noc, nr, nc);
	//��CCabs
	Cabs << <iDivUp(nor*noc, 128), 128 >> > (CC, CCabs, nor*noc);

	hipFree(CC); hipFree(kernc); hipFree(kernr);

}

__global__ void Cabs(hipfftComplex *CC, float *CCabs, unsigned int length)
{
	unsigned int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i < length)
	{
		CCabs[i] = sqrt(CC[i].x*CC[i].x + CC[i].y*CC[i].y);
	}
}

void dftregistration(IndexOffset &Ind, hipfftComplex *buf1ft, hipfftComplex *buf2ft, hipblasHandle_t handle,unsigned int height,unsigned int width, int usfac,int num)
{
	unsigned int nr = height;
	unsigned int nc = width;
	int beginNr = -floor((float)nr / 2);
	int endNr = ceil((float)nr / 2) - 1;
	int *Nr=new int[nr]();//!
	ifftshiftVec<int>(Nr, beginNr, endNr);

	int beginNc = -floor((float)nc / 2);
	int endNc = ceil((float)nc / 2) - 1;
	int *Nc = new int[nc]();
	ifftshiftVec<int>(Nc, beginNc, endNc);
	
	//���´���Ϊ usfac>1
	int nnr = 2 * nr; int nnc = 2 * nc;

	hipDoubleComplex *d_FTpad; hipMalloc(&d_FTpad, sizeof(hipDoubleComplex)*nnr*(nnc / 2 + 1)*num);//!
	hipDoubleComplex *d_dotTemp; hipMalloc(&d_dotTemp, sizeof(hipDoubleComplex)*nr*(nc / 2 + 1)*num);//!


	float rate = (float)(nnr*nnc) / (nr*nc);
	
	dot << <iDivUp((nc / 2 + 1)*nr, 256), 256 >> > (buf1ft, buf2ft, d_dotTemp, nr*(nc / 2 + 1), num, rate);//buf1ft.conj(buf2ft)

	FTpad(d_FTpad, d_dotTemp, nr, (nc / 2 + 1), nnr, (nnc / 2 + 1), num);//prepare for IFFT2



	dim3 block(16, 16);
	dim3 grid(iDivUp(nnc, block.x), iDivUp(nnr, block.y));
	hipDoubleComplex *d_CC; hipMalloc(&d_CC, sizeof(hipDoubleComplex)*nnr*nnc*num);//!
	for (int i = 0;i < num;++i)
	{
		getIn<hipDoubleComplex> << <grid, block >> > (d_FTpad + i * nnr*(nnc / 2 + 1), d_CC + i * nnr*nnc, nnr, nnc);
	}

	double *d_CCAbs;hipMalloc(&d_CCAbs, sizeof(double)*nnr*nnc*num);//!
	ifft2(d_CC, d_CC, nnr, nnc, num);//ifft2

	if (true)
	{
		hipFree(d_FTpad);
		hipFree(d_dotTemp);
	}

	Cabs << <iDivUp(nnr*nnc*num, 256), 256 >> > (d_CC, d_CCAbs, nnr*nnc*num);


	//getMaxIdx(Ind, d_CC, nnr, nnc, num);
	int maxInd = 0;
	int row_shift = 0; int col_shift = 0;

	beginNr = -floor((float)nr);
	endNr = ceil((float)nr) - 1;
	int *Nr2 = new int[nr*2]();//!
	ifftshiftVec<int>(Nr2, beginNr, endNr);

	beginNc = -floor((float)nc);
	endNc = ceil((float)nc) - 1;
	int *Nc2 = new int[nc*2]();
	ifftshiftVec<int>(Nc2, beginNc, endNc);

	for (int i = 0; i < num; ++i)
	{
		hipblasIdamax(handle, nnr*nnc, d_CCAbs + i * nnr*nnc, 1, &maxInd);
		row_shift = (maxInd - 1) / nnc;
		col_shift = (maxInd - 1) % nnc;
		//std::cout << col_shift << std::endl;
		//CCmax�ƺ�û�õ�
		row_shift = Nr2[row_shift];
		col_shift = Nc2[col_shift];
		Ind.indexX[i] = (float)col_shift / 2;
		Ind.indexY[i] = (float)row_shift / 2;
	}

	if (true)
	{
		delete[] Nr;
		delete[] Nc;
		delete[] Nr2;
		delete[] Nc2;
		hipFree(d_CC);
		hipFree(d_CCAbs);
	}

	//���´���Ϊusfac>2
	if (usfac > 2)
	{
		float dftshift = floor(ceil(usfac*1.5f) / 2);
		int noc = ceil(usfac*1.5); int nor = ceil(usfac*1.5);
		float roff = 0, coff = 0;
		//������Ҷ�������һ��
		hipfftComplex *in; hipMalloc(&in, sizeof(hipfftComplex)*height*width);
		double *CCabs; hipMalloc(&CCabs, sizeof(double)*noc*nor*num);
		for (int i = 0; i < num; ++i)
		{
			Ind.indexY[i] = round(Ind.indexY[i] * usfac) / usfac;
			Ind.indexX[i] = round(Ind.indexX[i] * usfac) / usfac;
			roff = dftshift - Ind.indexY[i] * usfac;
			coff = dftshift - Ind.indexX[i] * usfac;
			getDftupsIn(buf1ft, buf2ft + i * height*(width / 2 + 1), in, nr, nc);
			dftups(CCabs + i * nor*noc, in,handle, nor, noc, usfac, roff, coff, nr, nc);
			//��CC���ֵ
			hipblasIdamax(handle, nor*noc, CCabs + i * nor*noc, 1, &maxInd);
			row_shift = (maxInd-1) / noc;//75
			col_shift = (maxInd-1) % noc;//76
			row_shift = row_shift - dftshift ;
			col_shift = col_shift - dftshift ;
			Ind.indexY[i] = Ind.indexY[i] + (float)row_shift / usfac;
			Ind.indexX[i] = Ind.indexX[i] + (float)col_shift / usfac;
			//std::cout << i + 1 << "  ��:" << Ind.indexY[i] << "  ��:" << Ind.indexX[i] << std::endl;
		}
		if (true)
		{
			hipFree(in);
			hipFree(CCabs);
		}
	}

}

__global__ void leftMove(float *d_input, float *d_output,float x, unsigned int height, unsigned int width)
{
	unsigned int ix = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y*blockDim.y;
	if (ix < width&&iy < height)
	{
		unsigned int i = ix + iy * width;
		if (ix == 0)
		{
			d_output[i] = d_input[i] - d_input[i] * x;
		}
		else
		{
			d_output[i] = d_input[i] - (d_input[i] - d_input[ix - 1 + iy * width])*x;
		}
	}
}

__global__ void rightMove(float *d_input, float *d_output, float x, unsigned int height, unsigned int width)
{
	unsigned int ix = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y*blockDim.y;
	if (ix < width&&iy < height)
	{
		unsigned int i = ix + iy * width;
		if (ix == width - 1)
		{
			d_output[i] = d_input[i] - d_input[i] * x;
		}
		else
		{
			d_output[i] = d_input[i] - (d_input[ix + 1 + iy * width] - d_input[i])*x;
		}
	}
}

__global__ void downMove(float *d_input, float *d_output, float y, unsigned int height, unsigned int width)
{
	unsigned int ix = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y*blockDim.y;
	if (ix < width&&iy < height)
	{
		unsigned int i = ix + iy * width;
		if (iy == 0)
		{
			d_output[i] = d_input[i] - d_input[i] * y;
		}
		else
		{
			d_output[i] = d_input[i] - (d_input[i] - d_input[ix + (iy - 1)*width])*y;
		}
	}
}

__global__ void upMove(float *d_input, float *d_output, float y, unsigned int height, unsigned int width)
{
	unsigned int ix = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y*blockDim.y;
	if (ix < width&&iy < height)
	{
		unsigned int i = ix + iy * width;
		if (iy == height - 1)
		{
			d_output[i] = d_input[i] - d_input[i] * y;
		}
		else
		{
			d_output[i] = d_input[i] - (d_input[ix + (iy + 1)*width] - d_input[i])*y;
		}
	}
}

void bilinear(float *d_ImageIn, float x, float y, unsigned int oHeight, unsigned int oWidth)
{
	if (x == 0 && y == 0)
	{
		return;
	}
	float *temp; hipMalloc(&temp, sizeof(float)*oHeight*oWidth);
	dim3 block(16, 16);
	dim3 grid(iDivUp(oWidth, block.x), iDivUp(oHeight, block.y));
	//x����
	if (x != 0)
	{
		if (x > 0)//����  ��ǰ=��ǰ-���
		{
			leftMove << <grid, block >> > (d_ImageIn, temp, x, oHeight, oWidth);
		}
		else//���� ��ǰ=�ұ�-��ǰ
		{
			rightMove << <grid, block >> > (d_ImageIn, temp, x, oHeight, oWidth);
		}
		hipMemcpy(d_ImageIn, temp, sizeof(float)*oHeight*oWidth, hipMemcpyDeviceToDevice);
	}
	//y����
	if (y != 0)
	{
		if (y > 0)//���� ��ǰ��=��ǰ��-��һ��
		{
			downMove << <grid, block >> > (d_ImageIn, temp, y, oHeight, oWidth);
		}
		else//���� ��ǰ��=��һ��-��ǰ��
		{
			upMove << <grid, block >> > (d_ImageIn, temp, y, oHeight, oWidth);
		}
		hipMemcpy(d_ImageIn, temp, sizeof(float)*oHeight*oWidth, hipMemcpyDeviceToDevice);
	}

	hipFree(temp);

}

void translate(float *d_ImageIn, int x, int y, unsigned int oHeight, unsigned int oWidth)
{
	//����ƽ��
	if (x == 0 && y == 0)
	{
		return;
	}
	float *temp; hipMalloc(&temp, sizeof(float)*oHeight*oWidth);
	hipMemset(temp, 0, sizeof(float)*oHeight*oWidth);
	//y����ƽ��
	if (y != 0)
	{
		int absy = abs(y);
		if (y > 0)
			hipMemcpy(temp + absy * oWidth, d_ImageIn, sizeof(float)*(oHeight - absy)*oWidth, hipMemcpyDeviceToDevice);
		if (y < 0)
			hipMemcpy(temp, d_ImageIn + absy * oWidth, sizeof(float)*(oHeight - absy)*oWidth, hipMemcpyDeviceToDevice);
		if (x != 0)
		{
			hipMemset(d_ImageIn, 0, sizeof(float)*oHeight*oWidth);
			int absx = abs(x);
			for (int i = 0; i < oHeight; ++i)
			{
				if (x > 0)
					hipMemcpy(d_ImageIn + i * oWidth + absx, temp + i * oWidth, sizeof(float)*(oWidth - absx), hipMemcpyDeviceToDevice);
				if (x < 0)
					hipMemcpy(d_ImageIn + i * oWidth, temp + i * oWidth + absx, sizeof(float)*(oWidth - absx), hipMemcpyDeviceToDevice);
			}
		}
		else
		{
			hipMemcpy(d_ImageIn, temp, sizeof(float)*oHeight*oWidth, hipMemcpyDeviceToDevice);
		}
	}
	//x����ƽ��
	if (x != 0)
	{
		int absx = abs(x);
		for (int i = 0; i < oHeight; ++i)
		{
			if (x > 0)
				hipMemcpy(temp + i * oWidth + absx, d_ImageIn + i * oWidth, sizeof(float)*(oWidth - absx), hipMemcpyDeviceToDevice);
			if (x < 0)
				hipMemcpy(temp + i * oWidth, d_ImageIn + i * oWidth + absx, sizeof(float)*(oWidth - absx), hipMemcpyDeviceToDevice);
		}
		//y����ƽ��
		if (y != 0)
		{
			hipMemset(d_ImageIn, 0, sizeof(float)*oHeight*oWidth);
			int absy = abs(y);
			if (y > 0)
				hipMemcpy(d_ImageIn + absy * oWidth, temp, sizeof(float)*(oHeight - absy)*oWidth, hipMemcpyDeviceToDevice);
			if (y < 0)
				hipMemcpy(d_ImageIn, temp + absy * oWidth, sizeof(float)*(oHeight - absy)*oWidth, hipMemcpyDeviceToDevice);
		}
		else
		{
			hipMemcpy(d_ImageIn, temp, sizeof(float)*oHeight*oWidth, hipMemcpyDeviceToDevice);
		}
	}
	hipFree(temp);
}

void imtranslate(float *d_ImageIn,float x, float y, unsigned int oHeight, unsigned int oWidth)
{
	//��һ�Ų����ƶ�
	if (x == 0 && y == 0)
	{
		return;
	}
	else
	{
		float decimalX = 0; float decimalY = 0;
		decimalX = x - (int)x;//��
		decimalY = y - (int)y;//��
		//����λƽ��
		translate(d_ImageIn, (int)x, (int)y, oHeight, oWidth);
		//С��λ����˫���Բ�ֵ(�����㷨���̰ٶȲ�ѯ)
		bilinear(d_ImageIn, decimalX, decimalY, oHeight, oWidth);
	}
}

__global__ void getOutputImage(float *d_ImageIn, float *d_ImageOut, int startPointX, int startPointY,
	unsigned int inputHeight, unsigned int inputWidth, unsigned int outputHeight, unsigned int outputWidth)
{
	unsigned int ix = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y*blockDim.y;
	if (ix < outputWidth&&iy < outputHeight)
	{
		d_ImageOut[ix + iy * outputWidth] = d_ImageIn[startPointX + ix + (startPointY + iy)*inputWidth];
	}
}

void getArea(float *d_ImageIn, float *d_ImageOut, int startPointX, int startPointY,
	unsigned int inputHeight, unsigned int inputWidth, unsigned int outputHeight, unsigned int outputWidth)
{
	dim3 block(16, 16);
	dim3 grid(iDivUp(outputWidth, block.x), iDivUp(outputHeight, block.y));
	getOutputImage << <grid, block >> > (d_ImageIn, d_ImageOut, startPointX - 1, startPointY - 1, inputHeight, inputWidth, outputHeight, outputWidth);
}


/*
	�������ܣ�ͼ��ƥ��
	ImageIn��ͼ������
	ImageOut��ͼ�����
	d_fixedFFT����һ��ͼ(ģ��)��ά����Ҷ�仯���
	oHeight��ԭʼͼ��� oWidth��ԭʼͼ���
	nHeight�����ͼ��� nWidth�����ͼ���
	runImageNum������ƥ���ͼ������
*/
void registration(float *ImageIn, float *ImageOut,hipfftComplex *d_fixedFFT, unsigned int oHeight, unsigned int oWidth, unsigned int nHeight, unsigned int nWidth, unsigned int runImageNum)
{
	int startPointX = round((float)(oWidth - nWidth) / 2);//��
	int startPointY = round((float)(oHeight - nHeight) / 2);//��

	//float *d_fixed; hipMalloc(&d_fixed, sizeof(float)*oHeight*oWidth);//!
	//hipMemcpy(d_fixed, ImageIn, sizeof(float)*oHeight*oWidth, hipMemcpyHostToDevice);

	float *d_ImageIn; hipMalloc(&d_ImageIn, sizeof(float)*oHeight*oWidth * runImageNum);//!
	hipMemcpy(d_ImageIn, ImageIn, sizeof(float)*oHeight*oWidth*runImageNum, hipMemcpyHostToDevice);

	//�������
	//��ά����Ҷ���ȡһ�뼴�ɣ��ԳƵ�
	hipfftComplex *d_ImageInFFT; hipMalloc(&d_ImageInFFT, sizeof(hipfftComplex)*oHeight*(oWidth / 2 + 1)*runImageNum);//!
	fft2(d_ImageIn, d_ImageInFFT, oHeight, oWidth, runImageNum);


	hipblasHandle_t handle;
	hipblasCreate(&handle);
	IndexOffset ind(runImageNum);
	dftregistration(ind, d_fixedFFT, d_ImageInFFT,handle, oHeight, oWidth, 100, runImageNum);//�������ƫ����������ؾ����㷨�ٶȲ�ѯ��
	hipblasDestroy(handle);

	for (int i = 0;i < runImageNum;++i) {
		std::cout <<"indY:  "<<ind.indexY[i] <<"   indX: "<<ind.indexX[i] << std::endl;
	}
	std::cout << "===============================next Patch==============================" << std::endl;
	if (true)		
	{
		if (d_ImageInFFT != NULL) { hipFree(d_ImageInFFT); d_ImageInFFT = NULL; }
	}


	float *d_ImageOut; hipMalloc(&d_ImageOut, sizeof(float)*nHeight*nWidth*runImageNum);
	//getArea(d_fixed, d_ImageOut, startPointX, startPointY, oHeight, oWidth, nHeight, nWidth);
	for (int i = 0; i < runImageNum; ++i)
	{
		imtranslate(d_ImageIn + i * oHeight*oWidth, ind.indexX[i], ind.indexY[i], oHeight, oWidth);//ͨ��ƫ�����ƶ�ͼ��
		getArea(d_ImageIn + i * oHeight*oWidth, d_ImageOut + i * nHeight*nWidth, startPointX, startPointY, oHeight, oWidth, nHeight, nWidth);//��ȡͼ��
	}
	hipMemcpy(ImageOut, d_ImageOut, sizeof(float)*nHeight*nWidth*runImageNum, hipMemcpyDeviceToHost);

	//if (d_fixed != NULL)hipFree(d_fixed); d_fixed = NULL;
	if (d_ImageIn != NULL) { hipFree(d_ImageIn); d_ImageIn = NULL; }
	if (d_ImageOut != NULL) { hipFree(d_ImageOut); d_ImageOut = NULL; }
}

/*
	�������ܣ�ͼ������ν���ƥ��
	ImageIn��ͼ������
	ImageOut��ͼ�����
	oHeight��ԭʼͼ��� oWidth��ԭʼͼ���
	nHeight�����ͼ��� nWidth�����ͼ���
	num��ͼ��������
	ImageNumPerBatch��ÿ�����ε�ͼ������
*/
extern "C" void registrationGPU(float *ImageIn, float *ImageOut, unsigned int oHeight, unsigned int oWidth, unsigned int nHeight, unsigned int nWidth, unsigned int num,int ImageNumPerBatch)
{
	int startPointX = round((float)(oWidth - nWidth) / 2);//��
	int startPointY = round((float)(oHeight - nHeight) / 2);//��

	float *d_fixed; hipMalloc(&d_fixed, sizeof(float)*oHeight*oWidth);//!
	hipMemcpy(d_fixed, ImageIn, sizeof(float)*oHeight*oWidth, hipMemcpyHostToDevice);

	hipfftComplex *d_fixedFFT; hipMalloc(&d_fixedFFT, sizeof(hipfftComplex)*oHeight*(oWidth / 2 + 1));//!
	fft2(d_fixed, d_fixedFFT, oHeight, oWidth, 1);

	//��һ�Ų��ý���ƥ��
	float *d_ImageOutfirst; hipMalloc(&d_ImageOutfirst, sizeof(float)*nHeight*nWidth);
	getArea(d_fixed, d_ImageOutfirst, startPointX, startPointY, oHeight, oWidth, nHeight, nWidth);
	hipMemcpy(ImageOut, d_ImageOutfirst, sizeof(float)*nHeight*nWidth, hipMemcpyDeviceToHost);

	//ʣ�µ�ƥ��ͼ 
	//int ImageNumPerBatch = 7;
	int batch = ceil((float)(num - 1) / ImageNumPerBatch);//5

	//���ڼ�����ڴ����ޣ�������Ҫ����Ӧ�����ν��д���
	//��������ͼ���ܴ�СΪ30�ţ���ÿ����(ImageNumPerBatch)Ϊ7�š�����Ҫ5�δ���ǰ4�ζ�Ϊ7�ţ����һ��1�š�7+7+7+7+1=30-1����һ�Ų���Ҫƥ�䣬��ģ�塣
	for (int i = 0; i < batch; ++i)
	{
		int runImageNum = 0;
		if ((i + 1)*ImageNumPerBatch <= num - 1)//����һ��ImageNumPerBatch��batch
		{
			runImageNum = ImageNumPerBatch;
		}
		else//����һ��ImageNumPerBatch��batch
		{
			runImageNum = num - 1 - i * ImageNumPerBatch;
		}
		registration(ImageIn + (i*ImageNumPerBatch + 1)*oHeight*oWidth, ImageOut + (i*ImageNumPerBatch + 1)*nHeight*nWidth,d_fixedFFT,
			oHeight, oWidth, nHeight, nWidth, runImageNum);
	}

	if (d_fixed != NULL) { hipFree(d_fixed); d_fixed = NULL; }
	if (d_fixedFFT != NULL) { hipFree(d_fixedFFT); d_fixedFFT = NULL; }
	if (d_ImageOutfirst != NULL) { hipFree(d_ImageOutfirst); d_ImageOutfirst = NULL; }

}